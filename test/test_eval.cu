#include "hip/hip_runtime.h"
#include <iostream>

#include "symbolicmath.h"

using namespace symbolicmath;

using V0 = RuntimeValue<0>;
using V1 = RuntimeValue<1>;

__global__ void test_kernel()
{
	double d = Add<V0,V1>::type::eval(1.,2.);
	printf( "%f\n", d );
}

int main()
{
	double d = Add<V0,V1>::type::eval(1.,2.);
	std::cout << d << std::endl;
	
	test_kernel<<<1,1>>>();
	hipDeviceSynchronize();
}
