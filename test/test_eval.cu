#include "hip/hip_runtime.h"
#include <iostream>

#include "symbolicmath.h"

using namespace symbolicmath;

using V0 = RuntimeValue<0>;
using V1 = RuntimeValue<1>;

__global__ void test_kernel()
{
	double d = Add<V0,V1>::type::eval(1.,2.);
	printf( "%f\n", d );
	
	using type = Real<Trace<MatrixMult<PauliMatrix<1>::type,MatrixMult<PauliMatrix<2>::type, SU2::type>::type>::type>::type>::type;
	d = type::eval( 1,2,3,4 );
	printf( "trace Matrix (expected: 4) =  %f\n", d );
}

int main()
{
	double d = Add<V0,V1>::type::eval(1.,2.);
	std::cout << d << std::endl;
	
	test_kernel<<<1,1>>>();
	hipDeviceSynchronize();
}
